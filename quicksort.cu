#include "hip/hip_runtime.h"
/*
 * bitonic_sort.cu
 *
 */

#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"
#include "cuda_utils.h"

typedef struct QuickSort_args
{
    uint left;
    uint right;
} QuickSort_args;

__global__ static void CUDA_QuickSortGlobal(int* __restrict__ values,
                                            const uint left, const uint right,
                                            QuickSort_args* __restrict__ args_out)
{
    uint i = left;
    uint lt = left;
    uint gt = right;
    const int v = values[i];
    int v_i = v;

    while (i <= gt) {
        if (v_i < v) {
            values[i] = values[lt];
            values[lt++] = v_i;
            v_i = values[++i];
        } else if (v_i > v) {
            int temp = values[gt];
            values[i] = temp;
            values[gt--] = v_i;
            v_i = temp;
        } else {
            v_i = values[++i];
        }
    }

    args_out[0].left = left;
    args_out[0].right = lt > 0 ? lt - 1 : 0;
    args_out[1].left = gt + 1;
    args_out[1].right = right;
}

__global__ static void CUDA_QuickSortShared(int* __restrict__ values,
                                            const uint left, const uint right,
                                            QuickSort_args* __restrict__ args_out)
{
    uint i, lt, gt, v_i;
    extern __shared__ uint shared[];

    shared[TID] = values[TID+left];
    __syncthreads();

    if (TID == 0) {
        i = 0;
        lt = 0;
        gt = right - left;
        v_i = shared[i];
        const int pivot = v_i;

        while (i <= gt) {
            if (v_i < pivot) {
                shared[i] = shared[lt];
                shared[lt++] = v_i;
                v_i = shared[++i];
            } else if (v_i > pivot) {
                int temp = shared[gt];
                shared[i] = temp;
                shared[gt--] = v_i;
                v_i = temp;
            } else {
                v_i = shared[++i];
            }
        }
    }

    __syncthreads();
    values[TID+left] = shared[TID];

    if (TID == 0) {
        args_out[0].right = lt > 0 ? lt + left - 1 : 0;
        args_out[1].left = gt + left + 1;

    } else if (TID == TDIM - 1) {
        args_out[0].left = left;
        args_out[1].right = right;
    }
}

__host__ void inline QuickSort(int* d_mem_values,
                               const uint N)
{
    QuickSort_args *h_args_out, *d_args_out;
    hipStream_t *streams;

    h_args_out = (QuickSort_args*) malloc(sizeof(QuickSort_args));
    h_args_out->left = 0;
    h_args_out->right = N-1;

    uint i = 1;
    while(i > 0) {
        uint sorted = 0;
        streams = (hipStream_t*) malloc(sizeof(hipStream_t) * i);
        gpuErrchk( hipMalloc(&d_args_out, sizeof(QuickSort_args) * i * 2) );

        for (int j = 0; j < i; ++j) {
            kdim v = get_kdim(h_args_out[j].right - h_args_out[j].left + 1);
            gpuErrchk( hipStreamCreate(streams+j) );
            if (h_args_out[j].left < h_args_out[j].right) {
                if (N <= MAX_THREADS) {
                    CUDA_QuickSortShared<<<1, v.num_threads, v.num_threads*sizeof(int), streams[j]>>>(d_mem_values, h_args_out[j].left, h_args_out[j].right, d_args_out+(j<<1));
                } else {
                    CUDA_QuickSortGlobal<<<1, 1, 0, streams[j]>>>(d_mem_values, h_args_out[j].left, h_args_out[j].right, d_args_out+(j<<1));
                }
                ++sorted;
            }
        }

        if (sorted > 0) {
            QuickSort_args *h_args_out_new = (QuickSort_args*) malloc(sizeof(QuickSort_args) * sorted * 2);
            sorted = 0;

            for (int j = 0; j < i; ++j) {
                if (h_args_out[j].left < h_args_out[j].right) {
                    gpuErrchk( hipMemcpyAsync (h_args_out_new+((sorted++)<<1), d_args_out+(j<<1), sizeof(QuickSort_args) << 1, hipMemcpyDeviceToHost, streams[j]) );
                }
            }

            free(h_args_out);
            h_args_out = h_args_out_new;
        }

        hipDeviceSynchronize();
        hipFree(d_args_out);
        for (int j = 0; j < i; ++j)
            hipStreamDestroy(streams[j]);

        i = sorted*2;
        //printf("recurse: %d\n", i);
    }
    free(h_args_out);
}

// program main
int main(int argc, char** argv)
{
    void *h_mem, *d_mem_values;
    size_t min_size = 1024UL; //1kB
    size_t max_size = 1024UL*1024UL;//*1024UL*256UL; //256MB

    h_mem = malloc(max_size);
    assert(h_mem != NULL);
    gpuErrchk( hipMalloc(&d_mem_values, max_size) );

    srand(time(NULL));

    for(size_t size = min_size; size <= max_size; size <<= 1) {
        size_t N = size/sizeof(int);
        init_values_int((int*) h_mem, N);
        //printf("before %ld %s\n", N, is_int_array_sorted((int*) h_mem, N, false) ? "true":"false");
        //print_int_array((int*) h_mem, N);

        copy_to_device_time(d_mem_values, h_mem, size);
        hipDeviceSynchronize();

        QuickSort((int*) d_mem_values, N);
        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );

        copy_to_host_time(h_mem, d_mem_values, size);
        hipDeviceSynchronize();

        printf("after %ld %s\n", N, is_int_array_sorted((int*) h_mem, N, false) ? "true":"false");
        //print_int_array((int*) h_mem, N);
    }

    hipFree(d_mem_values);
    free(h_mem);

    return 0;
}
