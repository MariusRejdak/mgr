#include "hip/hip_runtime.h"
/*
 * bitonic_sort.cu
 *
 */

#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"
#include "cuda_utils.h"


__global__ static void CUDA_BitonicSort(int* __restrict__ values,
                                        const uint j, const uint k)
{
	const uint idx = TDIM * BID + TID;
	const uint ixj = idx^j;

	if (ixj > idx) {
		const uint v_idx = values[idx];
		const uint v_ixj = values[ixj];

		if ((idx&k) ? (v_idx < v_ixj) : (v_idx > v_ixj))
		{
			values[idx] = v_ixj;
			values[ixj] = v_idx;
		}
	}
}

__host__ void inline BitonicSort(int* d_mem, const uint N)
{
	kdim v = get_kdim(N);

	for (uint k = 2; k <= N; k <<= 1) {
		for (uint j = k >> 1; j > 0; j >>= 1) {
			CUDA_BitonicSort<<<v.dim_blocks, v.num_threads>>>(d_mem, j, k);
			hipDeviceSynchronize();
		}
	}
}

// program main
int main(int argc, char** argv) {
	void *h_mem, *d_mem;
	size_t min_size = 1024UL; //1kB
	size_t max_size = 1024UL*1024UL*256UL; //256MB

	h_mem = malloc(max_size);
	assert(h_mem != NULL);
	gpuErrchk(hipMalloc(&d_mem, max_size));

	srand(time(NULL));

	for(size_t size = min_size; size <= max_size; size <<= 1) {
		size_t N = size/sizeof(int);
		init_values_int((int*) h_mem, N);

		copy_to_device_time(d_mem, h_mem, size);
		hipDeviceSynchronize();

		BitonicSort((int*) d_mem, N);
		gpuErrchk( hipPeekAtLastError() );

		copy_to_host_time(h_mem, d_mem, size);
		hipDeviceSynchronize();

		printf("after %ld %s\n", N, is_int_array_sorted((int*) h_mem, N, false) ? "true":"false");
	}

	hipFree(d_mem);
	free(h_mem);

	return 0;
}
