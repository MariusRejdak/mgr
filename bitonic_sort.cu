#include "hip/hip_runtime.h"
/*
 * bitonic_sort.cu
 *
 */

#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"

typedef unsigned int uint;
typedef unsigned short ushort;

#define MAX_THREADS 512UL
#define MAX_DIM 32768UL

__global__ static void Bitonic_Sort(int* __restrict__ values, uint j, uint k, uint N) {
	const uint idx = gridDim.x * blockDim.x * blockIdx.y
					 + blockDim.x * blockIdx.x
					 + threadIdx.x;

	if (idx < N) {
		register const uint ixj = idx^j;
		register const uint v_idx = values[idx];
		register const uint v_ixj = values[ixj];

		if (ixj > idx) {
			if ((idx&k) == 0 && v_idx > values[ixj]) {
				values[idx] = v_ixj;
				values[ixj] = v_idx;
			}
			if ((idx&k) != 0 && values[idx] < values[ixj]) {
				values[idx] = v_ixj;
				values[ixj] = v_idx;
			}
		}
	}
}

// program main
int main(int argc, char** argv) {
	void *h_mem, *d_mem;
	size_t min_size = 1024UL; //1kB
	size_t max_size = 1024UL*1024UL*256UL; //256MB

	h_mem = malloc(max_size);
	assert(h_mem != NULL);
	gpuErrchk(hipMalloc(&d_mem, max_size));

	//srand(time(NULL));

	for(size_t size = min_size; size <= max_size; size <<= 1) {
		size_t N = size/sizeof(int);
		init_values_int((int*) h_mem, N);

		copy_to_device_time(d_mem, h_mem, size);
		hipDeviceSynchronize();

		for (uint k = 2; k <= N; k <<= 1) {
			for (uint j = k >> 1; j > 0; j >>= 1) {
				if (N <= MAX_THREADS) {
					Bitonic_Sort<<<1, N>>>((int*) d_mem, j, k, N);
				}
				else if(N <= MAX_DIM*MAX_THREADS) {
					dim3 blocks(N/MAX_THREADS);
					Bitonic_Sort<<<blocks, MAX_THREADS>>>((int*) d_mem, j, k, N);
				}
				else {
					dim3 blocks(MAX_DIM, N/MAX_THREADS/MAX_DIM + 1);
					Bitonic_Sort<<<blocks, MAX_THREADS>>>((int*) d_mem, j, k, N);
				}
				hipDeviceSynchronize();
			}
		}

		copy_to_host_time(h_mem, d_mem, size);
		hipDeviceSynchronize();

		printf("after %ld %s\n", N, is_int_array_sorted((int*) h_mem, N, false) ? "true":"false");
	}

	hipFree(d_mem);
	free(h_mem);

	return 0;
}
