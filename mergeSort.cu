#include "hip/hip_runtime.h"
/*
 * mergeSort.cu
 *
 */

#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"
#include "cuda_utils.h"

__global__ static void CUDA_MergeSortSmall(Element* __restrict__ values,
                                           const int32_t iteration)
{
    extern __shared__ Element shared_values[];
    const int32_t idx = TDIM * BID + TID;
    const int32_t srcMergeSize = 1 << iteration;
    Element* shared_a = shared_values;
    Element* shared_b = shared_values + srcMergeSize;
    Element* shared_out = shared_values + (srcMergeSize << 1);

    shared_values[TID] = values[idx];
    __syncthreads();

    if (TID == 0) {
        int32_t a = 0;
        int32_t b = 0;
        Element v_a = shared_a[a];
        Element v_b = shared_b[b];

        while (a + b < srcMergeSize) {
            if (b >= srcMergeSize || (a < srcMergeSize && v_a.k < v_b.k)) {
                shared_out[a + b] = v_a;
                v_a = shared_a[++a];
            } else {
                shared_out[a + b] = v_b;
                v_b = shared_b[++b];
            }
        }
    } else if (TID == TDIM-1) {
        int32_t a = srcMergeSize - 1;
        int32_t b = srcMergeSize - 1;
        Element v_a = shared_a[a];
        Element v_b = shared_b[b];

        while (a + b + 1 >= srcMergeSize) {
            if (b < 0 || (a >= 0 && v_a.k >= v_b.k)) {
                shared_out[a + b + 1] = v_a;
                v_a = --a > 0 ? shared_a[a] : v_a;
            } else {
                shared_out[a + b + 1] = v_b;
                v_b = --b > 0 ? shared_b[b] : v_b;
            }
        }
    }

    __syncthreads();
    values[idx] = shared_out[TID];
}


__global__ static void CUDA_MergeSortShared(Element* __restrict__ values,
                                            const int32_t iteration)
{
    extern __shared__ Element shared_values[];
    const int32_t idx = TDIM * BID + TID;
    Element* shared_a = shared_values;
    Element* shared_b = shared_values + (1 << iteration);
    Element* shared_out = shared_values + (2 << iteration);

    shared_a[TID] = values[idx];
    shared_b[TID] = values[idx + (1 << iteration)];
    __syncthreads();

    int32_t a = TID & ~(32 - 1);
    int32_t a_end = a + 32;
    int32_t b = a;
    int32_t b_end = a_end;

    if (a > 0) {
        const Key a_min = shared_a[a].k;
        while (b > 0 && a_min <= shared_b[b].k) b -= 32;
        while (b < TDIM-1 && a_min > shared_b[b].k) ++b;
    }
    if (a_end < TDIM) {
        const Key a_next_min = shared_a[a_end].k;
        while (b_end < TDIM && a_next_min > shared_b[b_end-1].k) b_end += 32;
        while (b_end > 0 && a_next_min <= shared_b[b_end-1].k) --b_end;
    }

    __syncthreads();

    if ((TID & 32 - 1) == 0) {
        Element v_a = shared_a[a];
        Element v_b = shared_b[b];

        while (a < a_end || b < b_end) {
            if (b >= b_end || (a < a_end && v_a.k < v_b.k)) {
                shared_out[a + b] = v_a;
                v_a = shared_a[++a];
            } else {
                shared_out[a + b] = v_b;
                v_b = shared_b[++b];
            }
        }
    }

    __syncthreads();
    values[idx << 1] = shared_out[TID << 1];
    values[(idx << 1) + 1] = shared_out[(TID << 1) + 1];
}

__global__ static void CUDA_MergeSortGlobal(Element* __restrict__ values,
                                            Element* __restrict__ values_sorted,
                                            const int32_t iteration,
                                            const int32_t N)
{
    /*extern __shared__ Element shared_values[];
    const int32_t idx = TDIM * BID + TID;

    values += idx*/
}

__host__ void inline MergeSort(Element** d_mem_values,
                               Element** d_mem_sorted,
                               const int32_t N)
{
    for (int32_t i = 0; (1 << i) < N; ++i) {

        if (i <= 6) {
            kdim v = get_kdim_nt(N, (2 << i));
            CUDA_MergeSortSmall<<<v.dim_blocks, v.num_threads, v.num_threads*sizeof(Element) << 1>>>(*d_mem_values, i);
        } else if ((1 << i) <= MAX_THREADS) {
            kdim v = get_kdim_nt(N/2, (1 << i));
            CUDA_MergeSortShared<<<v.dim_blocks, v.num_threads, v.num_threads*sizeof(Element) << 2>>>(*d_mem_values, i);
        }
        else {
            /*kdim v = get_kdim_b(N/i);
            CUDA_MergeSortGlobal<<<v.dim_blocks, v.num_threads>>>(*d_mem_values, *d_mem_sorted, i, N);
            swap((void**)d_mem_values, (void**)d_mem_sorted);*/
        }

        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );
    }

    swap((void**)d_mem_values, (void**)d_mem_sorted);
}

// program main
int main(int argc, char** argv)
{
    void *h_mem, *d_mem_values, *d_mem_sorted;

    h_mem = malloc(MAX_SIZE);
    assert(h_mem != NULL);
    gpuErrchk( hipMalloc(&d_mem_values, MAX_SIZE) );
    gpuErrchk( hipMalloc(&d_mem_sorted, MAX_SIZE) );

    srand(time(NULL));

    for(int32_t size = MIN_SIZE; size <= MAX_SIZE; size <<= 1) {
        int32_t N = size/sizeof(Element);
        init_values((Element*) h_mem, N);

        copy_to_device_time(d_mem_values, h_mem, size);
        hipDeviceSynchronize();

        MergeSort((Element**) &d_mem_values, (Element**) &d_mem_sorted, N);
        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );

        copy_to_host_time(h_mem, d_mem_sorted, size);
        hipDeviceSynchronize();

        printf("after %ld %s\n", N, is_int_array_sorted((Element*) h_mem, N, false) ? "true":"false");
    }

    hipFree(d_mem_values);
    hipFree(d_mem_sorted);
    free(h_mem);

    return 0;
}
