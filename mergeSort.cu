#include "hip/hip_runtime.h"
/*
 * mergeSort.cu
 *
 */

#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"
#include "cuda_utils.h"


__global__ static void CUDA_MergeSortShared(Element* __restrict__ values,
                                            const int32_t dstMergeSize)
{
    extern __shared__ Element shared_values[];
    const int32_t idx = TDIM * BID + TID;
    const int32_t srcMergeSize = dstMergeSize>>1;
    Element* shared_a = shared_values;
    Element* shared_b = shared_values+srcMergeSize;
    Element* shared_out = shared_values+dstMergeSize;

    shared_values[TID] = values[idx];
    __syncthreads();

    if (TID == 0) {
        int32_t i = 0;
        int32_t a = 0;
        int32_t b = 0;
        Element v_a = shared_a[a];
        Element v_b = shared_b[b];

        while (i < srcMergeSize) {
            if (b >= srcMergeSize || (a < srcMergeSize && v_a.k < v_b.k)) {
                shared_out[i++] = v_a;
                v_a = shared_a[++a];
            } else {
                shared_out[i++] = v_b;
                v_b = shared_b[++b];
            }
        }
    } else if (TID == TDIM-1) {
        int32_t i = dstMergeSize-1;
        int32_t a = srcMergeSize-1;
        int32_t b = srcMergeSize-1;
        Element v_a = shared_a[a];
        Element v_b = shared_b[b];

        while (i >= srcMergeSize) {
            if (b < 0 || (a > 0 && v_a.k >= v_b.k)) {
                shared_out[i--] = v_a;
                v_a = a > 0 ? shared_a[--a] : v_a;
            } else {
                shared_out[i--] = v_b;
                v_b = b > 0 ? shared_b[--b] : v_b;
            }
        }
    }

    __syncthreads();
    values[idx] = shared_out[TID];
}

__global__ static void CUDA_MergeSortGlobal(Element* __restrict__ values,
                                            Element* __restrict__ values_sorted,
                                            const int32_t iteration,
                                            const int32_t N)
{
    /*const int32_t lb_idx = TDIM * BID * 2;
    const int32_t idx = lb_idx + TID;

    const int32_t srcMergeSize = 1 << iteration; //2^iteration
    const int32_t dstMergeSize = srcMergeSize << 1; //2^(iteration+1)

    const int32_t lb_num = srcMergeSize / TDIM;

    const int32_t srcMergeIdA = (idx >> iteration) << iteration; // start, end = srcMergeIdA+srcMergeSize-1
    const int32_t srcMergeIdB = srcMergeIdA + srcMergeSize; // start, end = srcMergeIdB+srcMergeSize-1
    */

}

__host__ void inline MergeSort(Element** d_mem_values,
                               Element** d_mem_sorted,
                               const int32_t N)
{
    for (int32_t i = 0; (1 << i) < N; ++i) {

        if ((2 << i) <= MAX_THREADS) {
            kdim v = get_kdim_nt(N, (2 << i));
            CUDA_MergeSortShared<<<v.dim_blocks, v.num_threads, v.num_threads*sizeof(Element) << 1>>>(*d_mem_values, (2 << i));
        }
        else {
            kdim v = get_kdim_b(N/i);
            CUDA_MergeSortGlobal<<<v.dim_blocks, v.num_threads>>>(*d_mem_values, *d_mem_sorted, i, N);
            swap((void**)d_mem_values, (void**)d_mem_sorted);
        }

        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );
    }

    swap((void**)d_mem_values, (void**)d_mem_sorted);
}

// program main
int main(int argc, char** argv)
{
    void *h_mem, *d_mem_values, *d_mem_sorted;

    h_mem = malloc(MAX_SIZE);
    assert(h_mem != NULL);
    gpuErrchk( hipMalloc(&d_mem_values, MAX_SIZE) );
    gpuErrchk( hipMalloc(&d_mem_sorted, MAX_SIZE) );

    srand(time(NULL));

    for(int32_t size = MIN_SIZE; size <= MAX_SIZE; size <<= 1) {
        int32_t N = size/sizeof(Element);
        init_values((Element*) h_mem, N);

        copy_to_device_time(d_mem_values, h_mem, size);
        hipDeviceSynchronize();

        MergeSort((Element**) &d_mem_values, (Element**) &d_mem_sorted, N);
        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );

        copy_to_host_time(h_mem, d_mem_sorted, size);
        hipDeviceSynchronize();

        printf("after %ld %s\n", N, is_int_array_sorted((Element*) h_mem, N, false) ? "true":"false");
    }

    hipFree(d_mem_values);
    hipFree(d_mem_sorted);
    free(h_mem);

    return 0;
}
