#include "hip/hip_runtime.h"
/*
 * bitonic_sort.cu
 *
 */

#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"
#include "cuda_utils.h"


__global__ static void CUDA_MergeSortShared(int* __restrict__ values,
                                            const uint dstMergeSize)
{
    extern __shared__ int shared_values[];
    const uint idx = TDIM * BID + TID;
    const uint srcMergeSize = dstMergeSize>>1;
    int* shared_a = shared_values;
    int* shared_b = shared_values+srcMergeSize;
    int* shared_out = shared_values+dstMergeSize;

    shared_values[TID] = values[idx];
    __syncthreads();

    if (TID == 0) {
        uint i = 0;
        uint a = 0;
        uint b = 0;
        int v_a = shared_a[a];
        int v_b = shared_b[b];

        while (i < srcMergeSize) {
            if (b >= srcMergeSize || (a < srcMergeSize && v_a < v_b)) {
                shared_out[i++] = v_a;
                v_a = shared_a[++a];
            } else {
                shared_out[i++] = v_b;
                v_b = shared_b[++b];
            }
        }
    } else if (TID == TDIM-1) {
        uint i = dstMergeSize-1;
        uint a = srcMergeSize-1;
        uint b = srcMergeSize-1;
        int v_a = shared_a[a];
        int v_b = shared_b[b];

        while (i >= srcMergeSize) {
            if (b > srcMergeSize || (a < srcMergeSize && v_a >= v_b)) {
                shared_out[i--] = v_a;
                v_a = a > 0 ? shared_a[--a] : 0;
            } else {
                shared_out[i--] = v_b;
                v_b = b > 0 ? shared_b[--b] : 0;
            }
        }
    }

    __syncthreads();
    values[idx] = shared_out[TID];
}

__global__ static void CUDA_MergeSortGlobal(int* __restrict__ values,
                                            int* __restrict__ values_sorted,
                                            const uint iteration,
                                            const uint N)
{
    const uint lb_idx = TDIM * BID * 2;
    const uint idx = lb_idx + TID;

    const uint srcMergeSize = 1 << iteration; //2^iteration
    const uint dstMergeSize = srcMergeSize << 1; //2^(iteration+1)

    const uint lb_num = srcMergeSize / TDIM;

    const uint srcMergeIdA = (idx >> iteration) << iteration; // start, end = srcMergeIdA+srcMergeSize-1
    const uint srcMergeIdB = srcMergeIdA + srcMergeSize; // start, end = srcMergeIdB+srcMergeSize-1


}

__host__ void inline MergeSort(int** d_mem_values,
                               int** d_mem_sorted,
                               const uint N)
{
    for (uint i = 0; (1 << i) < N; ++i) {

        if ((2 << i) <= MAX_THREADS) {
            kdim v = get_kdim_nt(N, (2 << i));
            CUDA_MergeSortShared<<<v.dim_blocks, v.num_threads, v.num_threads*sizeof(int) << 1>>>(*d_mem_values, (2 << i));
        }
        else {
            //kdim v = get_kdim_b(N/i);
            CUDA_MergeSortGlobal<<<v.dim_blocks, v.num_threads>>>(*d_mem_values, *d_mem_sorted, i, N);
            //swap((void**)d_mem_values, (void**)d_mem_sorted);
        }

        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );
    }

    swap((void**)d_mem_values, (void**)d_mem_sorted);
}

// program main
int main(int argc, char** argv)
{
    void *h_mem, *d_mem_values, *d_mem_sorted;
    size_t min_size = 1024UL; //1kB
    size_t max_size = 1024UL*1024UL*256UL; //256MB

    h_mem = malloc(max_size);
    assert(h_mem != NULL);
    gpuErrchk( hipMalloc(&d_mem_values, max_size) );
    gpuErrchk( hipMalloc(&d_mem_sorted, max_size) );

    srand(time(NULL));

    for(size_t size = min_size; size <= max_size; size <<= 1) {
        size_t N = size/sizeof(int);
        init_values_int((int*) h_mem, N);

        copy_to_device_time(d_mem_values, h_mem, size);
        hipDeviceSynchronize();

        MergeSort((int**) &d_mem_values, (int**) &d_mem_sorted, N);
        hipDeviceSynchronize();
        gpuErrchk( hipPeekAtLastError() );

        copy_to_host_time(h_mem, d_mem_sorted, size);
        hipDeviceSynchronize();

        printf("after %ld %s\n", N, is_int_array_sorted((int*) h_mem, N, false) ? "true":"false");
    }

    hipFree(d_mem_values);
    hipFree(d_mem_sorted);
    free(h_mem);

    return 0;
}
