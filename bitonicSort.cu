#include "hip/hip_runtime.h"
/*
 * bitonicSort.cu
 * Author: Marius Rejdak
 */

#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"
#include "cuda_utils.h"

/*
 * Sortowanie bitoniczne w pamięci globalnej
 */
__global__ static void CUDA_BitonicSort_Global(
    Element* __restrict__ values, //Słowo kluczowe __restrict__ wyłącza optymalizacje kompilatora dla tej zmiennej
    const int32_t j,
    const int32_t k)
{
    // Aby zachować synchronizację kolejne kernele są wywoływane synchronicznie
    const int32_t idx = TDIM * BID + TID;
    const int32_t ixj = idx^j;

    if (ixj > idx) {
        // Zmienne lokalne (optymalizowane przez kompilator do rejestrów)
        // dla ograniczenia ilości odwołań do pamięci globalnej
        const Element v_idx = values[idx];
        const Element v_ixj = values[ixj];

        // Porównanie i zamiana
        if ((idx&k) ? (v_idx.k < v_ixj.k) : (v_idx.k > v_ixj.k)) {
            values[idx] = v_ixj;
            values[ixj] = v_idx;
        }
    }
}

/*
 * Sortowanie bitoniczne przeprowadzane w pamięci współdzielonej
 */
__global__ static void CUDA_BitonicSort_Shared(Element* __restrict__ values)
{
    extern __shared__ Element shared_values[];

    // Kopiowanie do pamięci współdzielonej
    shared_values[TID] = values[TID];

    // Oczekiwanie na zakończenie kopiowania
    __syncthreads();

    // Pętla sortująca
    for (int32_t k = 2; k <= TDIM; k <<= 1) {
        for (int32_t j = k >> 1; j > 0; j >>= 1) {
            const int32_t ixj = TID^j;

            if (ixj > TID) {
                // Zmienne lokalne (optymalizowane przez kompilator do rejestrów)
                // dla ograniczenia ilości odwołań do pamięci współdzielonej
                const Element v_idx = shared_values[TID];
                const Element v_ixj = shared_values[ixj];

                // Porównanie i zamiana
                if ((TID&k) ? (v_idx.k < v_ixj.k) : (v_idx.k > v_ixj.k)) {
                    shared_values[TID] = v_ixj;
                    shared_values[ixj] = v_idx;
                }
            }

            // Synchronizacja globalna
            __syncthreads();
        }
    }

    values[TID] = shared_values[TID];
}

/*
 * Wywołania funkcji kernel
 */
__host__ void inline BitonicSort(Element* d_mem, const int32_t N)
{
    kdim v = get_kdim(N);

    if (v.num_blocks == 1) {
        // Sortowanie można przeprowadzić w całości w pamięci współdzielonej
        CUDA_BitonicSort_Shared<<<v.dim_blocks, v.num_threads, v.num_threads * sizeof(Element)>>>(d_mem);
    } else {
        // Sortowanie należy przeprowadzić w pamięci globalnej
        for (int32_t k = 2; k <= N; k <<= 1) {
            for (int32_t j = k >> 1; j > 0; j >>= 1) {
                CUDA_BitonicSort_Global<<<v.dim_blocks, v.num_threads>>>(d_mem, j, k);
                hipDeviceSynchronize();
            }
        }
    }
}

// program main
int main(int argc, char** argv)
{
    void *h_mem, *d_mem;

    h_mem = malloc(MAX_SIZE);
    assert(h_mem != NULL);
    gpuErrchk(hipMalloc(&d_mem, MAX_SIZE));

    srand(time(NULL));

    printf("Bitonic sort\n");
    printf("%s,%s,%ld,%ld\n", "size", "time", CLOCKS_PER_SEC, sizeof(Element));

    for(int32_t size = MIN_SIZE; size <= MAX_SIZE; size <<= 1) {
        int32_t N = size/sizeof(Element);
        clock_t t1, t2, t_sum = 0;

        for (int i = 0; i < NUM_PASSES; ++i) {
            init_values((Element*) h_mem, N);

            copy_to_device_time(d_mem, h_mem, size);
            hipDeviceSynchronize();

            t1 = clock();
            BitonicSort((Element*) d_mem, N);
            hipDeviceSynchronize();
            t2 = clock();
            t_sum += t2 - t1;
            gpuErrchk( hipPeekAtLastError() );

            copy_to_host_time(h_mem, d_mem, size);
            hipDeviceSynchronize();

            assert(is_int_array_sorted((Element*) h_mem, N, false));
        }
        t_sum /= NUM_PASSES;

        printf("%ld,%ld\n", N, t_sum);
    }

    hipFree(d_mem);
    free(h_mem);

    return 0;
}
